#include "hip/hip_runtime.h"
/***************************************************************************
*1.将数据从主机内存数据复制到设备显存
*2.写好核函数
*3.CUDA编译器执行核函数 在GPU上完成计算操作
*4.把显存数据复制到主机内存
*5.释放显存空间
/***************************************************************************/

#include <stdio.h>
#include <stdlib.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>
#define DATA_SIZE 1048576
int data[DATA_SIZE];

//产生大量0-9之间的随机数
void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++) {
		number[i] = rand() % 10;
	}
}

//CUDA 初始化
bool InitCUDA()
{
	int count;
	//取得支持Cuda的装置的数目
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}

// __global__ 函数 (GPU上执行) 计算立方和 
//核函数是不可以有返回值类型的
__global__ static void sumOfSquares( int *num, int* result)
{
	int sum = 0;
	int i;
	for (i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i] * num[i];
	}
	*result = sum;
}

int main()
{
	//CUDA 初始化
	if (!InitCUDA()) {
		return 0;
	}
	//生成随机数
	GenerateNumbers(data, DATA_SIZE);
	/*把数据复制到显卡内存中*/
	int* gpudata, *result;
	//hipMalloc 取得一块显卡内存 ( 其中result用来存储计算结果 )
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	//hipMemcpy 将产生的随机数复制到显卡内存中 
	//hipMemcpyHostToDevice - 从内存复制到显卡内存
	//hipMemcpyDeviceToHost - 从显卡内存复制到内存
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	// 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
	sumOfSquares << <1, 1, 0 >> >(gpudata, result);
	/*把结果从显示芯片复制回主内存*/
	int sum;
	//hipMemcpy 将结果从显存中复制回内存
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	//Free
	hipFree(gpudata);
	hipFree(result);
	printf("GPUsum: %d \n", sum);
	sum = 0;
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i] * data[i];
	}
	printf("CPUsum: %d \n", sum);
	return 0;
}
