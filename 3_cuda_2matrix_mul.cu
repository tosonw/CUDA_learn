// CUDA编程之快速入门
// https://www.cnblogs.com/skyfsm/p/9673960.html

// 最后一个例子我们将计算一个更加复杂的任务，矩阵乘法
//
//回顾一下矩阵乘法：两矩阵相乘，左矩阵第一行乘以右矩阵第一列（分别相乘，第一个数乘第一个数），
// 乘完之后相加，即为结果的第一行第一列的数，依次往下算，直到计算完所有矩阵元素。
//
// 梳理一下CUDA求解矩阵乘法的思路：因为C=A×B，我们利用每个线程求解C矩阵每个(x, y)的元素，
// 每个线程载入A的一行和B的一列，遍历各自行列元素，对A、B对应的元素做一次乘法和一次加法。
//
// 从这个矩阵乘法任务可以看出，我们通过GPU进行并行计算的方式仅花费了0.5秒，
// 但是CPU串行计算方式却花费了7.6秒，计算速度提升了十多倍，可见并行计算的威力！

#define CPU

#ifdef GPU

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <sys/time.h>
#include <stdio.h>
#include <math.h>
#define Row 1024
#define Col 1024


__global__ void matrix_mul_gpu(int *M, int* N, int* P, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    int sum = 0;
    for(int k=0;k<width;k++)
    {
        int a = M[j*width+k];
        int b = N[k*width+i];
        sum += a*b;
    }
    P[j*width+i] = sum;
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );

    int *A = (int *)malloc(sizeof(int) * Row * Col);
    int *B = (int *)malloc(sizeof(int) * Row * Col);
    int *C = (int *)malloc(sizeof(int) * Row * Col);
    //malloc device memory
    int *d_dataA, *d_dataB, *d_dataC;
    cudaMalloc((void**)&d_dataA, sizeof(int) *Row*Col);
    cudaMalloc((void**)&d_dataB, sizeof(int) *Row*Col);
    cudaMalloc((void**)&d_dataC, sizeof(int) *Row*Col);
    //set value
    for (int i = 0; i < Row*Col; i++) {
        A[i] = 90;
        B[i] = 10;
    }

    cudaMemcpy(d_dataA, A, sizeof(int) * Row * Col, cudaMemcpyHostToDevice);
    cudaMemcpy(d_dataB, B, sizeof(int) * Row * Col, cudaMemcpyHostToDevice);
    dim3 threadPerBlock(16, 16);
    dim3 blockNumber((Col+threadPerBlock.x-1)/ threadPerBlock.x, (Row+threadPerBlock.y-1)/ threadPerBlock.y );
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    //拷贝计算数据-一级数据指针
    cudaMemcpy(C, d_dataC, sizeof(int) * Row * Col, cudaMemcpyDeviceToHost);

    //释放内存
    free(A);
    free(B);
    free(C);
    cudaFree(d_dataA);
    cudaFree(d_dataB);
    cudaFree(d_dataC);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse/1000);

    return 0;
}

//Block(16,16)   Grid(64,64).
//total time is 256 ms

#endif
#ifdef CPU


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <sys/time.h>
#define ROWS 1024
#define COLS 1024

using namespace std;

void matrix_mul_cpu(float* M, float* N, float* P, int width)
{
    for(int i=0;i<width;i++)
        for(int j=0;j<width;j++)
        {
            float sum = 0.0;
            for(int k=0;k<width;k++)
            {
                float a = M[i*width+k];
                float b = N[k*width+j];
                sum += a*b;
            }
            P[i*width+j] = sum;
        }
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    float *A, *B, *C;
    int total_size = ROWS*COLS*sizeof(float);
    A = (float*)malloc(total_size);
    B = (float*)malloc(total_size);
    C = (float*)malloc(total_size);

    //CPU一维数组初始化
    for(int i=0;i<ROWS*COLS;i++)
    {
        A[i] = 80.0;
        B[i] = 20.0;
    }

    matrix_mul_cpu(A, B, C, COLS);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;

    return 0;
}

//total time is 18087ms

#endif

